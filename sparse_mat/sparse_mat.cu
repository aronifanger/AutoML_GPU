#include <chrono>
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>

//#define cudaCheck(op) 						\\
//{								\\
//	int err;						\\
//	if (cudaSuccess != err = op) 				\\
//	{							\\
//		fprintf(stderr, "CUDA operation failed: %s\n",	\\
//				cudaGetErrorString(err));	\\
//		exit(EXIT_FAILURE);				\\
//	}							\\
//}

#define cudaCheck(op) op

__global__ void kernel(float *K, float *g1, float *M, int N, float k)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;
	if (i < N && j < N) M[i+j*N] = g1[N+i-j] * (K[i]+k) * (K[j]+k);
}

int main(int argc, char **argv)
{
	using namespace std::chrono;

	int N = 1000;
	if (argc == 2) {
		N = atoi(argv[1]);
	}
	printf("Using %d x %d\n", N, N); 

	float *M  = new float[N*N];
	float *K  = new float[N];
	float *g1 = new float[2*N];
	float k = 1.3;

	for (int i=0; i<N; i++)
		K[i]  = rand() / (float)RAND_MAX;
	for (int i=0; i<2*N; i++)
		g1[i] = rand() / (float)RAND_MAX;

	auto t0 = high_resolution_clock::now();

	float *d_M  = NULL;
	float *d_K  = NULL;
	float *d_g1 = NULL;

	cudaCheck(hipMalloc((void**)&d_M,  sizeof(float)*N*N));
	cudaCheck(hipMalloc((void**)&d_K,  sizeof(float)*N  ));
	cudaCheck(hipMalloc((void**)&d_g1, sizeof(float)*2*N));

	cudaCheck(hipMemcpy(d_M,   M, sizeof(float)*N*N, hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_K,   K, sizeof(float)*N  , hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_g1, g1, sizeof(float)*2*N, hipMemcpyHostToDevice));

	dim3 dimBlock(128); 
	dim3 dimGrid(N*N/dimBlock.x + (N*N % dimBlock.x ? 1 : 0));
	kernel<<<dimGrid,dimBlock>>>(K,g1,M,N,k);

	cudaCheck(hipGetLastError());
	cudaCheck(hipMemcpy(M, d_M, sizeof(float)*N*N, hipMemcpyDeviceToHost));
	cudaCheck(hipFree(d_M));
	cudaCheck(hipFree(d_K));
	cudaCheck(hipFree(d_g1));

	duration<float> t = high_resolution_clock::now() - t0;
	printf("GPU took %f seconds\n", t.count());

	delete[] M;
	delete[] K;
	delete[] g1;
	
	return 0;
}
